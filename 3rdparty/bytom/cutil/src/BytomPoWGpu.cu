#include "hip/hip_runtime.h"
#include "BytomPoW.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

void initMatVecGpu(BytomMatListGpu* matListGpu_int8, BytomMatList8* matList_int8) {
  hipMemcpy(matListGpu_int8->matVecGpu, matList_int8->matVec, 256 * 256 * 256, hipMemcpyHostToDevice);
}

__global__ void converInt32ToInt8_gpu(int32_t* a, int8_t* b) {
  int32_t data_i32 = a[blockIdx.x * blockDim.x + threadIdx.x];
  int8_t data_i8 = ((data_i32&0xFF)+ ((data_i32>>8)&0xFF))&0xFF;
  b[blockIdx.x * blockDim.x + threadIdx.x] = data_i8;
}

void core_mineBytom_gpu(
        std::vector<uint8_t> fourSeq[4],
        BytomMatListGpu* matListGpu_int8,
        uint32_t data[64],
        hipblasHandle_t handle) {

  Mat256x256i8 *idt=new Mat256x256i8;
  Mat256x256i8 *mat=new Mat256x256i8;
  Mat256x256i8 *tmp=new Mat256x256i8;
  Mat256x256i8 *res=new Mat256x256i8[4];
  idt->toIdentityMatrix();

  int8_t* devIdt_i8;
  int8_t* devTmp_i8;
  int32_t* devTmp_i32;
  hipMalloc ((void**)&devIdt_i8, 256*256*sizeof(*devIdt_i8));
  hipMalloc ((void**)&devTmp_i8, 256*256*sizeof(*devTmp_i8));
  hipMalloc ((void**)&devTmp_i32, 256*256*sizeof(*devTmp_i32));
  hipblasStatus_t stat = hipblasSetMatrix (256, 256, sizeof(*devIdt_i8), idt->d, 256, devIdt_i8, 256);
  const int alpha = 1;
  const int beta = 0;

  for(int k=0; k<4; k++) {
    for(int j=0; j<2; j++) {
      for(int i=0; i<32; i+=2) {
        if (j==0 && i==0)
          stat = hipblasGemmEx(handle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              256,
                              256,
                              256,
                              &alpha,
                              matListGpu_int8->at(fourSeq[k][i]),
                              HIP_R_8I,
                              256,
                              devIdt_i8,
                              HIP_R_8I,
                              256,
                              &beta,
                              devTmp_i32,
                              HIP_R_32I,
                              256,
                              HIP_R_32I,
                              HIPBLAS_GEMM_DEFAULT);
        else
          stat = hipblasGemmEx(handle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              256,
                              256,
                              256,
                              &alpha,
                              matListGpu_int8->at(fourSeq[k][i]),
                              HIP_R_8I,
                              256,
                              devTmp_i8,
                              HIP_R_8I,
                              256,
                              &beta,
                              devTmp_i32,
                              HIP_R_32I,
                              256,
                              HIP_R_32I,
                              HIPBLAS_GEMM_DEFAULT); 
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          std::cout<<"Fail to Run CuBlas GemmEx.1"<<std::endl;
          std::cout<<stat<<std::endl;
          std::cout<<"skip"<<std::endl;
          return;
          // exit(EXIT_FAILURE);
        }
        converInt32ToInt8_gpu<<<256, 256>>>(devTmp_i32, devTmp_i8);
        stat = hipblasGemmEx(handle,
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            256,
                            256,
                            256,
                            &alpha,
                            matListGpu_int8->at(fourSeq[k][i+1]),
                            HIP_R_8I,
                            256,
                            devTmp_i8,
                            HIP_R_8I,
                            256,
                            &beta,
                            devTmp_i32,
                            HIP_R_32I,
                            256,
                            HIP_R_32I,
                            HIPBLAS_GEMM_DEFAULT); 
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          std::cerr<<"Fail to Run CuBlas GemmEx.2"<<std::endl;
          exit(EXIT_FAILURE);
        }
        converInt32ToInt8_gpu<<<256, 256>>>(devTmp_i32, devTmp_i8);
      }
    }
    stat = hipblasGetMatrix (256, 256, sizeof(*devTmp_i8), devTmp_i8, 256, res[k].d, 256);
  }

  mat->add(res[0], res[1]);  
  tmp->add(*mat, res[2]);    
  mat->add(*tmp, res[3]);    

  Arr256x64i32 arr(*mat);
  arr.reduceFNV();           
  arr.fillWithD0(data);      
  
  delete idt;
  delete mat;
  delete tmp;
  delete[] res;

  hipFree(devIdt_i8);
  hipFree(devTmp_i8);
  hipFree(devTmp_i32);
  hipblasDestroy(handle);
}
